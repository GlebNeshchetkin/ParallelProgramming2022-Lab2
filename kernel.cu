#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>


const int N = 5;
__device__
const double delta_tau = 0.00, delta_t = 0.01;
#define PI 3.14159265358979323846
__device__
double p = 2000, m = 100, g = 10, v = 0;

__device__
void get_x1(double X2[5], double X0[5], double Ax) {
    X2[0] = (X0[0] + X0[2] * cos(3 * PI / 2 - X0[3]) - Ax);
}

__device__
void get_x2(double X2[5], double X0[5], double Bx) {
    X2[1] = (X0[1] + X0[2] * cos(3 * PI / 2 + X0[4]) - Bx);
}

__device__
void get_y(double* X2, double* X0, double Ay) {
    X2[2] = (X0[2] + X0[2] * sin(3 * PI / 2 - X0[3]) - Ay);
}

__device__
void get_f1(double* X2, double* X0, double C) {
    X2[3] = ((X0[3] + X0[4]) * X0[2] + (X0[1] - X0[0]) - C);
}

__device__
void get_f2(double X2[5], double X0[5], double By) {
    X2[4] = (X0[2] + X0[2] * sin(3 * PI / 2 + X0[4]) - By);
}

__global__
void parallel(double* X0, double* X1, double* X2, double Ax, double Ay, double Bx, double By, double C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    switch (i) {
    case 0:
        get_x1(X2, X0, Ax);
        X1[i] = X0[i] - delta_tau * X2[i];
        break;
    case 1:
        get_x2(X2, X0, Bx);
        X1[i] = X0[i] - delta_tau * X2[i];
        break;
    case 2:
        get_y(X2, X0, Ay);
        X1[i] = X0[i] - delta_tau * X2[i];
        break;
    case 3:
        get_f1(X2, X0, C);
        X1[i] = X0[i] - delta_tau * X2[i];
        break;
    case 4:
        get_f2(X2, X0, By);
        X1[i] = X0[i] - delta_tau * X2[i];
        break;
    }
}

__global__
void serial(double* X0, double* X1, double* X2, double Ax, double Ay, double Bx, double By, double C) {
    get_x1(X2, X0, Ax);
    get_x2(X2, X0, Bx);
    get_y(X2, X0, Ay);
    get_f1(X2, X0, C);
    get_f2(X2, X0, By);
    for (int i = 0; i < 5; i++) {
        X1[i] = X0[i] - delta_tau * X2[i];
    }
}

__global__
void parallelA(double* A0, double* A1, double x1, double x2) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("%f-%f-%f", A0[0], x1, x2);
    switch (i) {
    case 0:
        A1[0] = A0[0] + A0[1] * delta_t;
        break;
    case 1:
        A1[1] = A0[1] + (p * (x2 - x1) - m * g) / m * delta_t;;
        break;
    }
}

double Ax = -0.353, Ay = 0.3, Bx = 0.353, By = 0.3, C = 3 * PI / 8;

int main() {
    //FILE* ouf = fopen("output.txt", "w");

    double* x0, * x1, * x2, * d_x0, * d_x1, * d_x2, * a0, * a1, * d_a0, * d_a1;
    x0 = (double*)malloc(N * sizeof(double));
    x1 = (double*)malloc(N * sizeof(double));
    x2 = (double*)malloc(N * sizeof(double));
    a0 = (double*)malloc(2 * sizeof(double));
    a1 = (double*)malloc(2 * sizeof(double));

    hipMalloc(&d_x0, N * sizeof(double));
    hipMalloc(&d_x1, N * sizeof(double));
    hipMalloc(&d_x2, N * sizeof(double));
    hipMalloc(&d_a0, 2 * sizeof(double));
    hipMalloc(&d_a1, 2 * sizeof(double));

    double x0_init[] = { -0.1, 0.1, 0.0, 2.0, 2.0 };
    memcpy(x0, x0_init, N * sizeof(double));
    a0[0] = 0.3;
    a0[1] = 0;
    double time_sum = 0;
    double time_taken;
    double time_complete_sum = 0;
    hipMemcpy(d_a0, a0, 2 * sizeof(double), hipMemcpyHostToDevice);
    for (double t = 0; t <= 0.05; t += delta_t) {
        
        hipMemcpy(d_x0, x0, N * sizeof(double), hipMemcpyHostToDevice);
        time_sum = 0;
        for (int step = 0; step < 300000; ++step) {
            double flag = true;
            clock_t start = clock();
            //parallel<<<1, 5>>>(d_x0, d_x1, d_x2, Ax, Ay, Bx, By, C);
            serial << <1, 1 >> > (d_x0, d_x1, d_x2, Ax, Ay, Bx, By, C);
            clock_t end = clock();
            time_taken = ((double)(end - start)) / CLOCKS_PER_SEC;
            time_sum += time_taken;
            hipMemcpy(d_x0, d_x1, N * sizeof(double), hipMemcpyDeviceToDevice);
            hipMemcpy(x2, d_x2, N * sizeof(double), hipMemcpyDeviceToHost);
            for (int i = 0; i < 5; i++) { if (x2[i] > 0.0001) { flag = false; } }
            if (flag) { break; }
        }

        hipMemcpy(x1, d_x1, N * sizeof(double), hipMemcpyDeviceToHost);
        time_complete_sum += time_sum;
        printf("%f\n", time_sum);
        parallelA << <1, 2 >> > (d_a0, d_a1, x1[0], x1[1]);
        //hipMemcpy(a1, d_a1, 2 * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(d_a0, d_a1, 2 * sizeof(double), hipMemcpyDeviceToDevice);
        hipMemcpy(a1, d_a1, 2 * sizeof(double), hipMemcpyDeviceToHost);
        Ay = a1[0];
        By = a1[0];
        v = a1[1];

        for (int i = 0; i < 5; ++i) {
            printf("%f ", x1[i]);
        }
        printf("\n");
        //fprintf(ouf, "%f, %f, %f, %f, %f, %f\n", Ax, Ay, Bx, By, C, time_taken);
        //fflush(ouf);
    }
    printf("->%f\n", time_complete_sum/6);
    hipFree(d_x0);
    hipFree(d_x1);
    free(x0);
    free(x1);
}
